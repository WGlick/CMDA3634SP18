#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"


int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */

  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */
  FILE *file = fopen("public_key.txt","r");
  fscanf(file,"%u\n%u\n%u\n%u\n",&n,&p,&g,&h);
  fclose(file);

  // find the secret key
  if (x==0 || modExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    double startTime = clock();
    for (unsigned int i=0;i<p-1;i++) {
      if (modExp(g,i+1,p)==h) {
        printf("Secret key found! x = %u \n", i+1);
        x=i+1;
      } 
    }
    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  }

  /* Q3 After finding the secret key, decrypt the message */

  unsigned int Nchars;


  FILE *file2 = fopen("messages.txt","r");
  fscanf(file2,"%u %u\n",&Nchars,&Nints);
  unsigned int *Zmessage = (unsigned int *) malloc(Nints * sizeof(unsigned int)); 
  unsigned int *a = (unsigned int *) malloc(Nints*sizeof(unsigned int));
  unsigned char *message = (unsigned char *) malloc(Nints*sizeof(unsigned char));
    for (int i =0; i < Nints; i++) {
      fscanf(file2,"%u %u\n",&Zmessage[i], &a[i]);
  }
  fclose(file2);

  ElGamalDecrypt(Zmessage,a,Nints,p,x);

  convertZToString(Zmessage, Nints, message, Nchars);
  printf("%s \n",message);

  return 0;
}
